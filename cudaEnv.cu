#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <algorithm>
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>


#include <intrin.h>
#define SH_ASSERT( ExpectTrue ) \
	if( ( ExpectTrue ) == 0 )   \
	{                           \
		__debugbreak();         \
	}


struct splitmix64
{
	uint64_t x = 0; /* The state can be seeded with any value. */

	uint64_t next()
	{
		uint64_t z = ( x += 0x9e3779b97f4a7c15 );
		z = ( z ^ ( z >> 30 ) ) * 0xbf58476d1ce4e5b9;
		z = ( z ^ ( z >> 27 ) ) * 0x94d049bb133111eb;
		return z ^ ( z >> 31 );
	}
};

class CudaStopwatch
{
public:
	CudaStopwatch( hipStream_t stream )
	{
		m_stream = stream;
		hipEventCreateWithFlags( &m_start, hipEventDefault );
		hipEventCreateWithFlags( &m_stop, hipEventDefault );
	}
	~CudaStopwatch()
	{
		hipEventDestroy( m_start );
		hipEventDestroy( m_stop );
	}

	void start() { hipEventRecord( m_start, m_stream ); }
	void stop() { hipEventRecord( m_stop, m_stream ); }

	float getMs()
	{
		hipEventSynchronize( m_stop );
		float ms = 0;
		hipEventElapsedTime( &ms, m_start, m_stop );
		return ms;
	}

public:
	hipStream_t m_stream;
	hipEvent_t m_start;
	hipEvent_t m_stop;
};

int main()
{
	hipInit( 0 );

	hipDevice_t cuDevice;
	hipError_t res = hipDeviceGet( &cuDevice, 0 );
	if( res != hipSuccess )
	{
		printf( "cannot acquire device 0\n" );
		exit( 1 );
	}

	hipCtx_t cuContext;
	res = hipCtxCreate( &cuContext, 0, cuDevice );
	if( res != hipSuccess )
	{
		printf( "cannot create context\n" );
		exit( 1 );
	}

	hipStream_t stream;
	hipStreamCreateWithFlags( &stream, hipStreamDefault );

	std::vector<uint32_t> inputs( 160 * 1000 * 1000 );

	uint32_t* inputBuffer;
	uint32_t* outputBuffer;
	hipMalloc( (void**)&inputBuffer, inputs.size() * sizeof( uint32_t ) );
	hipMalloc( (void**)&outputBuffer, inputs.size() * sizeof( uint32_t ) );

	void* d_temp_storage = NULL;
	size_t temp_storage_bytes = 0;

	hipcub::DeviceRadixSort::SortKeys( d_temp_storage, temp_storage_bytes, inputBuffer, outputBuffer, inputs.size(), 0, 32, stream, false );
	hipMalloc( (void**)&d_temp_storage, temp_storage_bytes );

	splitmix64 rng;
	for (;;)
	{
		for( int i = 0; i < inputs.size(); i++ )
		{
			inputs[i] = rng.next() & 0xFFFFFFFF;
		}
		hipMemcpyHtoD( (hipDeviceptr_t)inputBuffer, inputs.data(), inputs.size() * sizeof( uint32_t ) );

		CudaStopwatch cudaSw( stream );
		cudaSw.start();

		hipError_t e = hipcub::DeviceRadixSort::SortKeys( d_temp_storage, temp_storage_bytes, inputBuffer, outputBuffer, inputs.size(), 0, 32, stream, false );

		cudaSw.stop();
		float ms = cudaSw.getMs();
		hipStreamSynchronize( stream );

		printf( "%f ms\n", ms );

		break;

		std::vector<uint32_t> outputs( inputs.size() );
		hipError_t r = hipMemcpyDtoH( outputs.data(), (hipDeviceptr_t)outputBuffer, inputs.size() * sizeof( uint32_t ) );
		for( int i = 0; i < outputs.size() - 1; i++ )
		{
			SH_ASSERT( outputs[i] <= outputs[i + 1] );
		}
		std::sort( inputs.begin(), inputs.end() );
		for( int i = 0; i < outputs.size(); i++ )
		{
			SH_ASSERT( outputs[i] == inputs[i] );
		}
	}
}